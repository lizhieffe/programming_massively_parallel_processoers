// This is the practice for MatMul in Chapter 02.
// It can run in LeetGPU playground.

#include <iostream>
#include <hip/hip_runtime.h>

__global__ void MatrixMulKernel(float* M, float* N, float* P, int Width) {
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    if ((row < Width) && (col < Width)) {
        float PValue = 0.0;
        for (int k = 0; k < Width; k++) {
            PValue += M[row * Width + k] * N[k * Width + col];
        }
        P[row * Width + col] = PValue;
    }
}

int WIDTH = 4;

int main() {
    float a_h[4][4] = {{1.0, 2.0, 3.0, 4.0}, {5.0,6.0, 7.0, 8.0}, {9.0, 10.0, 11.0, 12.0}, {13.0, 14.0, 15.0, 16.0}};
    float b_h[4][4] = {{1.0, 2.0, 3.0, 4.0}, {5.0,6.0, 7.0, 8.0}, {9.0, 10.0, 11.0, 12.0}, {13.0, 14.0, 15.0, 16.0}};
    float c_h[4][4] = {{0.0, 0.0, 0.0, 0.0}, {0.0, 0.0, 0.0, 0.0}, {0.0, 0.0, 0.0, 0.0}, {0.0, 0.0, 0.0, 0.0}};

    // debug
    for (int i = 0; i < WIDTH; i++) {
        for (int j = 0; j < WIDTH; j++) {
            printf("%f\n", a_h[i][j]);
        }
    }

    // Allocate device memory
    int size = WIDTH * WIDTH * sizeof(float);
    float *a_d, *b_d, *c_d;

    hipMalloc((void **)&a_d, size);
    hipMalloc((void **)&b_d, size);
    hipMalloc((void **)&c_d, size);

    hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice);

    // Call kernel
    // gridDim = 2x2, blockDim = 2x2
    MatrixMulKernel<<<dim3(2, 2, 1), dim3(2, 2, 1)>>>(a_d, b_d, c_d, WIDTH);

    // Copy from the device memory
    hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost);

    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);

    // Check result
    printf("\nResult is:\n");
    for (int i = 0; i < WIDTH; i++) {
        for (int j = 0; j < WIDTH; j++) {
            printf("%f\n", c_h[i][j]);
        }
    }
    
    return 0;
}
